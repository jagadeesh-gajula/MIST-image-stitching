/// ================================================================
/// 
/// Disclaimer:  IMPORTANT:  This software was developed at theNT
/// National Institute of Standards and Technology by employees of the
/// Federal Government in the course of their official duties.
/// Pursuant to title 17 Section 105 of the United States Code this
/// software is not subject to copyright protection and is in the
/// public domain.  This is an experimental system.  NIST assumes no
/// responsibility whatsoever for its use by other parties, and makes
/// no guarantees, expressed or implied, about its quality,
/// reliability, or any other characteristic.  We would appreciate
/// acknowledgement if the software is used.  This software can be
/// redistributed and/or modified freely provided that any derivative
/// works bear some notice that they are derived from it, and any
/// modified versions bear some notice that they have been modified.
/// 
/// ================================================================

// ================================================================
// 
// Author: Timothy Blattner
// Date:   Wed Nov 30 12:36:40 2011 EScufftDoubleComplex
// 
// Functions that execute on the graphics card for doing
// Vector computation.
// 
// ================================================================

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include<float.h>

#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

// ================================================================
__device__ double distance(int x1, int x2, int y1, int y2)
{
	return ((double(x1-x2))*(double(x1-x2)))+
			((double(y1-y2))*(double(y1-y2)));
}

__device__ bool checkDistance(int *maxesRow, 
		int *maxesCol, int nMax, 
		int curIdx, int width)
{
	int row = curIdx / width;
	int col = curIdx % width;
	int j;
	//double dist;
	for (j = 0; j < nMax; j++)
	{
			if (maxesRow[j] == row && maxesCol[j] == col)
  			return false;

		//dist = distance(maxesRow[j], row, maxesCol[j], col);

		//if (dist < MIN_DISTANCE)
		//	return false;


	}

	return true;
}

__device__ bool checkDistance(volatile int *maxesRow, 
		volatile int *maxesCol, int nMax, 
		int curIdx, int width)
{
	int row = curIdx / width;
	int col = curIdx % width;
	int j;
	//double dist;
	for (j = 0; j < nMax; j++)
	{

		if (maxesRow[j] == row && maxesCol[j] == col)
			return false;

//		dist = distance(maxesRow[j], row, maxesCol[j], col);

//		if (dist < MIN_DISTANCE)
//			return false;


	}

	return true;
}



extern "C"
	__global__ void
elt_prod_conj(hipfftDoubleComplex *fc, hipfftDoubleComplex * c1, 
		hipfftDoubleComplex * c2, int size)
{
	__shared__ hipfftDoubleComplex sfc[THREADS_PER_BLOCK];
	__shared__ hipfftDoubleComplex sc1[THREADS_PER_BLOCK];
	__shared__ hipfftDoubleComplex sc2[THREADS_PER_BLOCK];

	int idx = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

	if (idx >= size)
		return;

	sc1[threadIdx.x] = c1[idx];
	sc2[threadIdx.x] = c2[idx];

	__syncthreads();

	sfc[threadIdx.x] = hipCmul(sc1[threadIdx.x], hipConj(sc2[threadIdx.x]));

	double mag = hipCabs(sfc[threadIdx.x]);
	
	if (mag == 0 || isnan(mag))
	{
		mag = DBL_EPSILON;
		sfc[threadIdx.x].x = DBL_EPSILON;
	}
	
	fc[idx] = make_hipDoubleComplex(hipCreal(sfc[threadIdx.x]) / mag,
			hipCimag(sfc[threadIdx.x]) / mag);
}

extern "C"
	__global__ void
elt_prod_conj_v2(hipfftDoubleComplex *fc, hipfftDoubleComplex * c1, 
		hipfftDoubleComplex * c2, int size)
{
	__shared__ hipfftDoubleComplex sfc[THREADS_PER_BLOCK];

	int idx = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

	if (idx >= size)
		return;


	//cufftDoubleComplex fc_res;

	sfc[threadIdx.x] = hipCmul(c1[idx], hipConj(c2[idx]));

	__syncthreads();

	double mag;

	//  mag = sqrt(fc_res.x * fc_res.x + fc_res.y * fc_res.y);
	mag = sqrt(sfc[threadIdx.x].x * sfc[threadIdx.x].x +
			sfc[threadIdx.x].y * sfc[threadIdx.x].y);

	if (isnan(mag) || mag == 0)
	{
		mag = DBL_EPSILON; //cuCabs(sfc[threadIdx.x]);
		sfc[threadIdx.x].x = DBL_EPSILON;
	}

	
//	if (mag == 0)
//		mag = DBL_EPSILON;
	
	
	fc[idx] = make_hipDoubleComplex(sfc[threadIdx.x].x / mag,
			sfc[threadIdx.x].y / mag);
}

extern "C"
	__global__ void
elt_prod_conj_v3(hipfftDoubleComplex *fc, hipfftDoubleComplex * c1,
		hipfftDoubleComplex *c2, int size)
{
	int idx = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

	if (idx >= size)
		return;

	hipfftDoubleComplex _c1 = c1[idx];
	hipfftDoubleComplex _c2 = c2[idx];
	hipfftDoubleComplex _fc = hipCmul(_c1, hipConj(_c2));
	double mag = sqrt(_fc.x * _fc.x +
			_fc.y * _fc.y);

	if (isnan(mag) || mag == 0)
		mag = hipCabs(_fc);
	
	if (mag == 0)
		mag = DBL_EPSILON;
	
	fc[idx] = make_hipDoubleComplex(_fc.x / mag, _fc.y / mag);
}

extern "C"
	__global__ void
reduce_max_final(double *g_idata, double *g_odata, 
		int * max_idx, unsigned int n, int blockSize)
{
	__shared__ double sdata[THREADS_PER_BLOCK];
	__shared__ int idxData[THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;


	double myMax = 0.0;
	int myMaxIndex;

	while (i < n)
	{
		if (myMax < g_idata[i])
		{
			myMax = g_idata[i];
			myMaxIndex = max_idx[i];
		}

		if (i+blockSize < n)
		{
			if (myMax < g_idata[i+blockSize])
			{
				myMax = g_idata[i+blockSize];
				myMaxIndex = max_idx[i+blockSize];
			}
		}

		i += gridSize;
	}

	sdata[tid] = myMax;
	idxData[tid] = myMaxIndex;

	__syncthreads();

	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			if (myMax < sdata[tid + 256])
			{
				sdata[tid] = myMax = sdata[tid+256];
				idxData[tid] = idxData[tid+256];
			}
		}
		__syncthreads();
	}

	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			if (myMax < sdata[tid + 128])
			{
				sdata[tid] = myMax = sdata[tid+128];
				idxData[tid] = idxData[tid+128];
			}
		}
		__syncthreads();
	}

	if (blockSize >= 128)
	{
		if (tid <   64)
		{
			if(myMax < sdata[tid +   64])
			{
				sdata[tid] = myMax = sdata[tid+64];
				idxData[tid] = idxData[tid+64];
			}
		}
		__syncthreads();
	}

	volatile double *vdata = sdata;
	volatile int *vidxData = idxData;

	if (tid < 32)
	{
		if (blockSize >=  64)
			if (myMax < vdata[tid + 32])
			{
				vdata[tid] = myMax = vdata[tid+32];
				vidxData[tid] = vidxData[tid+32];
			}

		if (blockSize >=  32)
			if (myMax < vdata[tid + 16])
			{
				vdata[tid] = myMax = vdata[tid+16];
				vidxData[tid] = vidxData[tid+16];
			}

		if (blockSize >=  16)
			if (myMax < vdata[tid +  8])
			{
				vdata[tid] = myMax = vdata[tid+8];
				vidxData[tid] = vidxData[tid+8];
			}

		if (blockSize >=    8)
			if (myMax < vdata[tid +  4])
			{
				vdata[tid] = myMax = vdata[tid+4];
				vidxData[tid] = vidxData[tid+4];
			}

		if (blockSize >=    4)
			if (myMax < vdata[tid+2])
			{
				vdata[tid] = myMax = vdata[tid+2];
				vidxData[tid] = vidxData[tid+2];
			}

		if (blockSize >=    2)
			if (myMax < vdata[tid +  1])
			{
				vdata[tid] = myMax = vdata[tid+1];
				vidxData[tid] = vidxData[tid+1];
			}
		__syncthreads();
	}

	if (tid == 0)
	{
		g_odata[blockIdx.x] = sdata[0];
		max_idx[blockIdx.x] = idxData[0];
	}
}

extern "C"
	__global__ void
reduce_max_main(double *g_idata, double *g_odata, 
		int * max_idx, unsigned int n, int blockSize)
{
	__shared__ double sdata[THREADS_PER_BLOCK];
	__shared__ int idxData[THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize) + tid;
	unsigned int gridSize = blockSize*gridDim.x;


	double myMax = 0.0;
	int myMaxIndex;
	double val;

	while (i < n)
	{
		val = g_idata[i];
		if (myMax < val)
		{
			myMax = val;
			myMaxIndex = i;
		}

		if (i+blockSize < n)
		{
			val = g_idata[i+blockSize];
			if (myMax < val)
			{
				myMax = val;
				myMaxIndex = i+blockSize;
			}
		}

		i += gridSize;
	}

	sdata[tid] = myMax;
	idxData[tid] = myMaxIndex;

	__syncthreads();

	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			if (myMax < sdata[tid + 256])
			{
				sdata[tid] = myMax = sdata[tid+256];
				idxData[tid] = idxData[tid+256];
			}
		}
		__syncthreads();
	}

	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			if (myMax < sdata[tid + 128])
			{
				sdata[tid] = myMax = sdata[tid+128];
				idxData[tid] = idxData[tid+128];
			}
		}
		__syncthreads();
	}

	if (blockSize >= 128)
	{
		if (tid <   64)
		{
			if(myMax < sdata[tid +   64])
			{
				sdata[tid] = myMax = sdata[tid+64];
				idxData[tid] = idxData[tid+64];
			}
		}
		__syncthreads();
	}

	volatile double *vdata = sdata;
	volatile int *vidxData = idxData;

	if (tid < 32)
	{
		if (blockSize >=  64)
			if (myMax < vdata[tid + 32])
			{
				vdata[tid] = myMax = vdata[tid+32];
				vidxData[tid] = vidxData[tid+32];
			}

		if (blockSize >=  32)
			if (myMax < vdata[tid + 16])
			{
				vdata[tid] = myMax = vdata[tid+16];
				vidxData[tid] = vidxData[tid+16];
			}

		if (blockSize >=  16)
			if (myMax < vdata[tid +  8])
			{
				vdata[tid] = myMax = vdata[tid+8];
				vidxData[tid] = vidxData[tid+8];
			}

		if (blockSize >=    8)
			if (myMax < vdata[tid +  4])
			{
				vdata[tid] = myMax = vdata[tid+4];
				vidxData[tid] = vidxData[tid+4];
			}

		if (blockSize >=    4)
			if (myMax < vdata[tid+2])
			{
				vdata[tid] = myMax = vdata[tid+2];
				vidxData[tid] = vidxData[tid+2];
			}

		if (blockSize >=    2)
			if (myMax < vdata[tid +  1])
			{
				vdata[tid] = myMax = vdata[tid+1];
				vidxData[tid] = vidxData[tid+1];
			}
		__syncthreads();
	}

	if (tid == 0)
	{
		g_odata[blockIdx.x] = sdata[0];
		max_idx[blockIdx.x] = idxData[0];
	}
}


extern "C"
	__global__ void
reduce_max_filter_final(double *g_idata, double *g_odata, 
		int * max_idx, unsigned int n, unsigned int width, 
		int blockSize, 
		int *maxes, int nMax)
{
	__shared__ int smaxesRow[10];
	__shared__ int smaxesCol[10];
	__shared__ int smaxesVal[10];
	__shared__ double sdata[THREADS_PER_BLOCK];
	__shared__ int idxData[THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;

	if (tid < nMax)
	{
		smaxesVal[tid] = maxes[tid];
		smaxesRow[tid] = smaxesVal[tid] / width;		
		smaxesCol[tid] = smaxesVal[tid] % width;		
	}
	__syncthreads();

	double myMax = 0.0;
	int myMaxIndex;

	while (i < n)
	{
		if (myMax < g_idata[i])
		{
			if (checkDistance(smaxesRow, smaxesCol,
						nMax, max_idx[i], width))
			{
				myMax = g_idata[i];
				myMaxIndex = max_idx[i];
			}
		}

		if (i+blockSize < n)
		{
			if (myMax < g_idata[i+blockSize])
			{
				if (checkDistance(smaxesRow, smaxesCol, 
							nMax, 
							max_idx[i+blockSize], 
							width))
				{

					myMax = g_idata[i+blockSize];
					myMaxIndex = max_idx[i+blockSize];
				}
			}
		}

		i += gridSize;
	}

	sdata[tid] = myMax;
	idxData[tid] = myMaxIndex;

	__syncthreads();

	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			if (myMax < sdata[tid + 256])
			{
				if (checkDistance(smaxesRow, smaxesCol, 
							nMax, idxData[tid+256], 
							width))
				{
					sdata[tid] = myMax = sdata[tid+256];
					idxData[tid] = idxData[tid+256];
				}
			}
		}
		__syncthreads();
	}

	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			if (myMax < sdata[tid + 128])
			{
				if (checkDistance(smaxesRow, smaxesCol, 
							nMax, idxData[tid+128], 
							width))
				{
					sdata[tid] = myMax = sdata[tid+128];
					idxData[tid] = idxData[tid+128];
				}
			}
		}
		__syncthreads();
	}

	if (blockSize >= 128)
	{
		if (tid <   64)
		{
			if(myMax < sdata[tid +   64])
			{
				if (checkDistance(smaxesRow, smaxesCol, 
							nMax, idxData[tid+64], 
							width))
				{
					sdata[tid] = myMax = sdata[tid+64];
					idxData[tid] = idxData[tid+64];
				}
			}
		}
		__syncthreads();
	}

	volatile double *vdata = sdata;
	volatile int *vidxData = idxData;

	volatile int *vsmaxesRow = smaxesRow;
	volatile int *vsmaxesCol = smaxesCol;
	
	if (tid < 32)
	{
		if (blockSize >=  64)
			if (myMax < vdata[tid + 32])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+32], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+32];
					vidxData[tid] = vidxData[tid+32];
				}
			}

		if (blockSize >=  32)
			if (myMax < vdata[tid + 16])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+16], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+16];
					vidxData[tid] = vidxData[tid+16];
				}
			}

		if (blockSize >=  16)
			if (myMax < vdata[tid +  8])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+8], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+8];
					vidxData[tid] = vidxData[tid+8];
				}
			}

		if (blockSize >=    8)
			if (myMax < vdata[tid +  4])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+4], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+4];
					vidxData[tid] = vidxData[tid+4];
				}
			}

		if (blockSize >=    4)
			if (myMax < vdata[tid+2])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+2], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+2];
					vidxData[tid] = vidxData[tid+2];
				}
			}

		if (blockSize >=    2)
			if (myMax < vdata[tid +  1])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+1], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+1];
					vidxData[tid] = vidxData[tid+1];
				}

			}
		__syncthreads();
	}
	

	if (tid == 0)
	{
		g_odata[blockIdx.x] = sdata[0];
		max_idx[blockIdx.x] = idxData[0];

		if (gridDim.x == 1)
    		maxes[nMax] = idxData[0];
	}
}


extern "C"
	__global__ void
reduce_max_filter_main(double *g_idata, double *g_odata, 
		int * max_idx, unsigned int width, unsigned int height,
		int blockSize, 
		int *maxes, int nMax)
{
	__shared__ int smaxesRow[10];
	__shared__ int smaxesCol[10];
	__shared__ int smaxesVal[10];
	__shared__ double sdata[THREADS_PER_BLOCK];
	__shared__ int idxData[THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize) + tid;
	unsigned int gridSize = blockSize*gridDim.x;
	if (tid < nMax)
	{
		smaxesVal[tid] = maxes[tid];
		smaxesRow[tid] = smaxesVal[tid] / width;		
		smaxesCol[tid] = smaxesVal[tid] % width;		
	}
	__syncthreads();

	double myMax = -INFINITY;
	int myMaxIndex;
	double val;

	while (i < width * height)
	{
		val = g_idata[i];
		if (myMax < val)
		{
			// compute distance . . .
			if (checkDistance(smaxesRow, smaxesCol, 
						nMax, i, width))
			{
				myMax = val;
				myMaxIndex = i;
			}
		}

		if (i+blockSize < width * height)
		{
			val = g_idata[i+blockSize];
			if (myMax < val)
			{

				if (checkDistance(smaxesRow, smaxesCol, 
							nMax, i+blockSize, width))
				{
					myMax = val;
					myMaxIndex = i+blockSize;
				}
			}
		}

		i += gridSize;
	}

	sdata[tid] = myMax;
	idxData[tid] = myMaxIndex;

	__syncthreads();

	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			if (myMax < sdata[tid + 256])
			{
				if (checkDistance(smaxesRow, smaxesCol, 
							nMax, idxData[tid+256],
							width))
				{
					sdata[tid] = myMax = sdata[tid+256];
					idxData[tid] = idxData[tid+256];
				}
			}
		}
		__syncthreads();
	}

	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			if (myMax < sdata[tid + 128])
			{
				if (checkDistance(smaxesRow, smaxesCol, 
							nMax, idxData[tid+128],
							width))
				{
					sdata[tid] = myMax = sdata[tid+128];
					idxData[tid] = idxData[tid+128];
				}
			}
		}
		__syncthreads();
	}

	if (blockSize >= 128)
	{
		if (tid <   64)
		{
			if(myMax < sdata[tid +   64])
			{
				if (checkDistance(smaxesRow, smaxesCol, 
							nMax, idxData[tid+64], 
							width))
				{
					sdata[tid] = myMax = sdata[tid+64];
					idxData[tid] = idxData[tid+64];
				}
			}
		}
		__syncthreads();
	}

	volatile double *vdata = sdata;
	volatile int *vidxData = idxData;

	volatile int *vsmaxesRow = smaxesRow;
	volatile int *vsmaxesCol = smaxesCol;
	
	if (tid < 32)
	{
		if (blockSize >=  64)
			if (myMax < vdata[tid + 32])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+32], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+32];
					vidxData[tid] = vidxData[tid+32];
				}
			}

		if (blockSize >=  32)
			if (myMax < vdata[tid + 16])
			{

				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+16], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+16];
					vidxData[tid] = vidxData[tid+16];
				}
			}

		if (blockSize >=  16)
			if (myMax < vdata[tid +  8])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+8], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+8];
					vidxData[tid] = vidxData[tid+8];
				}
			}

		if (blockSize >=    8)
			if (myMax < vdata[tid +  4])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+4], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+4];
					vidxData[tid] = vidxData[tid+4];
				}
			}

		if (blockSize >=    4)
			if (myMax < vdata[tid+2])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+2], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+2];
					vidxData[tid] = vidxData[tid+2];
				}
			}

		if (blockSize >=    2)
			if (myMax < vdata[tid +  1])
			{
				if (checkDistance(vsmaxesRow, vsmaxesCol, 
							nMax, vidxData[tid+1], 
							width))
				{
					vdata[tid] = myMax = vdata[tid+1];
					vidxData[tid] = vidxData[tid+1];
				}
			}
		__syncthreads();
	}

	if (tid == 0)
	{
		g_odata[blockIdx.x] = sdata[0];
		max_idx[blockIdx.x] = idxData[0];

		if (gridDim.x == 1)
    		maxes[nMax] = idxData[0];
	}
}


// ================================================================
// ================================================================
// ================================================================
// ================================================================
// ======================= Float versions =========================
// ================================================================
// ================================================================
// ================================================================
// ================================================================

// ================================================================
__device__ float distancef(int x1, int x2, int y1, int y2)
{
	return ((float(x1-x2))*(float(x1-x2)))+
			((float(y1-y2))*(float(y1-y2)));
}

__device__ bool checkDistancef(int *maxesRow, 
		int *maxesCol, int nMax, 
		int curIdx, int width)
{
	int row = curIdx / width;
	int col = curIdx % width;
	int j;
	for (j = 0; j < nMax; j++)
	{
			if (maxesRow[j] == row && maxesCol[j] == col)
  			return false;

		//dist = distance(maxesRow[j], row, maxesCol[j], col);

		//if (dist < MIN_DISTANCE)
		//	return false;


	}

	return true;
}

__device__ bool checkDistancef(volatile int *maxesRow,
		volatile int *maxesCol, int nMax,
		int curIdx, int width)
{
	int row = curIdx / width;
	int col = curIdx % width;
	int j;
	for (j = 0; j < nMax; j++)
	{

		if (maxesRow[j] == row && maxesCol[j] == col)
			return false;


	}

	return true;
}

extern "C"
__global__ void
elt_prod_conjf(hipfftComplex *fc, hipfftComplex * c1,
		hipfftComplex * c2, int size)
{
	__shared__ hipfftComplex sfc[THREADS_PER_BLOCK];
	__shared__ hipfftComplex sc1[THREADS_PER_BLOCK];
	__shared__ hipfftComplex sc2[THREADS_PER_BLOCK];

	int idx = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

	if (idx >= size)
		return;

	sc1[threadIdx.x] = c1[idx];
	sc2[threadIdx.x] = c2[idx];

	__syncthreads();

	sfc[threadIdx.x] = hipCmulf(sc1[threadIdx.x], hipConjf(sc2[threadIdx.x]));

	float mag = hipCabsf(sfc[threadIdx.x]);

	if (mag == 0 || isnan(mag))
	{
		mag = FLT_EPSILON;
		sfc[threadIdx.x].x = FLT_EPSILON;
	}

	fc[idx] = make_hipComplex(hipCrealf(sfc[threadIdx.x]) / mag,
			hipCimagf(sfc[threadIdx.x]) / mag);
}

extern "C"
__global__ void
elt_prod_conj_v2f(hipfftComplex *fc, hipfftComplex * c1,
		hipfftComplex * c2, int size)
{
	__shared__ hipfftComplex sfc[THREADS_PER_BLOCK];

	int idx = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

	if (idx >= size)
		return;


	//cufftDoubleComplex fc_res;

	sfc[threadIdx.x] = hipCmulf(c1[idx], hipConjf(c2[idx]));

	__syncthreads();

	float mag;

	//  mag = sqrt(fc_res.x * fc_res.x + fc_res.y * fc_res.y);
	mag = sqrtf(sfc[threadIdx.x].x * sfc[threadIdx.x].x +
			sfc[threadIdx.x].y * sfc[threadIdx.x].y);

	if (isnan(mag) || mag == 0)
	{
		mag = FLT_EPSILON; //cuCabs(sfc[threadIdx.x]);
		sfc[threadIdx.x].x = FLT_EPSILON;
	}


//	if (mag == 0)
//		mag = DBL_EPSILON;


	fc[idx] = make_hipComplex(sfc[threadIdx.x].x / mag,
			sfc[threadIdx.x].y / mag);
}

extern "C"
__global__ void
elt_prod_conj_v3f(hipfftComplex *fc, hipfftComplex * c1,
		hipfftComplex *c2, int size)
{
	int idx = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

	if (idx >= size)
		return;

	hipfftComplex _c1 = c1[idx];
	hipfftComplex _c2 = c2[idx];
	hipfftComplex _fc = hipCmulf(_c1, hipConjf(_c2));
	float mag = sqrtf(_fc.x * _fc.x +
			_fc.y * _fc.y);

	if (isnan(mag) || mag == 0)
		mag = hipCabsf(_fc);

	if (mag == 0)
		mag = FLT_EPSILON;

	fc[idx] = make_hipComplex(_fc.x / mag, _fc.y / mag);
}

extern "C"
__global__ void
reduce_max_finalf(float *g_idata, float *g_odata,
		int * max_idx, unsigned int n, int blockSize)
{
	__shared__ float sdata[THREADS_PER_BLOCK];
	__shared__ int idxData[THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;


	float myMax = 0.0;
	int myMaxIndex;

	while (i < n)
	{
		if (myMax < g_idata[i])
		{
			myMax = g_idata[i];
			myMaxIndex = max_idx[i];
		}

		if (i+blockSize < n)
		{
			if (myMax < g_idata[i+blockSize])
			{
				myMax = g_idata[i+blockSize];
				myMaxIndex = max_idx[i+blockSize];
			}
		}

		i += gridSize;
	}

	sdata[tid] = myMax;
	idxData[tid] = myMaxIndex;

	__syncthreads();

	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			if (myMax < sdata[tid + 256])
			{
				sdata[tid] = myMax = sdata[tid+256];
				idxData[tid] = idxData[tid+256];
			}
		}
		__syncthreads();
	}

	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			if (myMax < sdata[tid + 128])
			{
				sdata[tid] = myMax = sdata[tid+128];
				idxData[tid] = idxData[tid+128];
			}
		}
		__syncthreads();
	}

	if (blockSize >= 128)
	{
		if (tid <   64)
		{
			if(myMax < sdata[tid +   64])
			{
				sdata[tid] = myMax = sdata[tid+64];
				idxData[tid] = idxData[tid+64];
			}
		}
		__syncthreads();
	}

	volatile float *vdata = sdata;
	volatile int *vidxData = idxData;

	if (tid < 32)
	{
		if (blockSize >=  64)
			if (myMax < vdata[tid + 32])
			{
				vdata[tid] = myMax = vdata[tid+32];
				vidxData[tid] = vidxData[tid+32];
			}

		if (blockSize >=  32)
			if (myMax < vdata[tid + 16])
			{
				vdata[tid] = myMax = vdata[tid+16];
				vidxData[tid] = vidxData[tid+16];
			}

		if (blockSize >=  16)
			if (myMax < vdata[tid +  8])
			{
				vdata[tid] = myMax = vdata[tid+8];
				vidxData[tid] = vidxData[tid+8];
			}

		if (blockSize >=    8)
			if (myMax < vdata[tid +  4])
			{
				vdata[tid] = myMax = vdata[tid+4];
				vidxData[tid] = vidxData[tid+4];
			}

		if (blockSize >=    4)
			if (myMax < vdata[tid+2])
			{
				vdata[tid] = myMax = vdata[tid+2];
				vidxData[tid] = vidxData[tid+2];
			}

		if (blockSize >=    2)
			if (myMax < vdata[tid +  1])
			{
				vdata[tid] = myMax = vdata[tid+1];
				vidxData[tid] = vidxData[tid+1];
			}
		__syncthreads();
	}

	if (tid == 0)
	{
		g_odata[blockIdx.x] = sdata[0];
		max_idx[blockIdx.x] = idxData[0];
	}
}

extern "C"
__global__ void
reduce_max_mainf(float *g_idata, float *g_odata,
		int * max_idx, unsigned int n, int blockSize)
{
	__shared__ float sdata[THREADS_PER_BLOCK];
	__shared__ int idxData[THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize) + tid;
	unsigned int gridSize = blockSize*gridDim.x;


	float myMax = 0.0;
	int myMaxIndex;
	float val;

	while (i < n)
	{
		val = g_idata[i];
		if (myMax < val)
		{
			myMax = val;
			myMaxIndex = i;
		}

		if (i+blockSize < n)
		{
			val = g_idata[i+blockSize];
			if (myMax < val)
			{
				myMax = val;
				myMaxIndex = i+blockSize;
			}
		}

		i += gridSize;
	}

	sdata[tid] = myMax;
	idxData[tid] = myMaxIndex;

	__syncthreads();

	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			if (myMax < sdata[tid + 256])
			{
				sdata[tid] = myMax = sdata[tid+256];
				idxData[tid] = idxData[tid+256];
			}
		}
		__syncthreads();
	}

	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			if (myMax < sdata[tid + 128])
			{
				sdata[tid] = myMax = sdata[tid+128];
				idxData[tid] = idxData[tid+128];
			}
		}
		__syncthreads();
	}

	if (blockSize >= 128)
	{
		if (tid <   64)
		{
			if(myMax < sdata[tid +   64])
			{
				sdata[tid] = myMax = sdata[tid+64];
				idxData[tid] = idxData[tid+64];
			}
		}
		__syncthreads();
	}

	volatile float *vdata = sdata;
	volatile int *vidxData = idxData;

	if (tid < 32)
	{
		if (blockSize >=  64)
			if (myMax < vdata[tid + 32])
			{
				vdata[tid] = myMax = vdata[tid+32];
				vidxData[tid] = vidxData[tid+32];
			}

		if (blockSize >=  32)
			if (myMax < vdata[tid + 16])
			{
				vdata[tid] = myMax = vdata[tid+16];
				vidxData[tid] = vidxData[tid+16];
			}

		if (blockSize >=  16)
			if (myMax < vdata[tid +  8])
			{
				vdata[tid] = myMax = vdata[tid+8];
				vidxData[tid] = vidxData[tid+8];
			}

		if (blockSize >=    8)
			if (myMax < vdata[tid +  4])
			{
				vdata[tid] = myMax = vdata[tid+4];
				vidxData[tid] = vidxData[tid+4];
			}

		if (blockSize >=    4)
			if (myMax < vdata[tid+2])
			{
				vdata[tid] = myMax = vdata[tid+2];
				vidxData[tid] = vidxData[tid+2];
			}

		if (blockSize >=    2)
			if (myMax < vdata[tid +  1])
			{
				vdata[tid] = myMax = vdata[tid+1];
				vidxData[tid] = vidxData[tid+1];
			}
		__syncthreads();
	}

	if (tid == 0)
	{
		g_odata[blockIdx.x] = sdata[0];
		max_idx[blockIdx.x] = idxData[0];
	}
}

extern "C"
__global__ void
reduce_max_filter_finalf(float *g_idata, float *g_odata,
		int * max_idx, unsigned int n, unsigned int width, 
		int blockSize, 
		int *maxes, int nMax)
{
	__shared__ int smaxesRow[10];
	__shared__ int smaxesCol[10];
	__shared__ int smaxesVal[10];
	__shared__ float sdata[THREADS_PER_BLOCK];
	__shared__ int idxData[THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;

	if (tid < nMax)
	{
		smaxesVal[tid] = maxes[tid];
		smaxesRow[tid] = smaxesVal[tid] / width;		
		smaxesCol[tid] = smaxesVal[tid] % width;		
	}
	__syncthreads();

	float myMax = 0.0;
	int myMaxIndex;

	while (i < n)
	{
		if (myMax < g_idata[i])
		{
			if (checkDistancef(smaxesRow, smaxesCol, 
					nMax, max_idx[i], width))
			{		
				myMax = g_idata[i];
				myMaxIndex = max_idx[i];
			}
		}

		if (i+blockSize < n)
		{
			if (myMax < g_idata[i+blockSize])
			{
				if (checkDistancef(smaxesRow, smaxesCol, 
						nMax,
						max_idx[i+blockSize],
						width))
				{

					myMax = g_idata[i+blockSize];
					myMaxIndex = max_idx[i+blockSize];
				}
			}
		}

		i += gridSize;
	}

	sdata[tid] = myMax;
	idxData[tid] = myMaxIndex;

	__syncthreads();

	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			if (myMax < sdata[tid + 256])
			{
				if (checkDistancef(smaxesRow, smaxesCol, 
						nMax, idxData[tid+256],
						width))
				{
					sdata[tid] = myMax = sdata[tid+256];
					idxData[tid] = idxData[tid+256];
				}
			}
		}
		__syncthreads();
	}

	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			if (myMax < sdata[tid + 128])
			{
				if (checkDistancef(smaxesRow, smaxesCol, 
						nMax, idxData[tid+128],
						width))
				{
					sdata[tid] = myMax = sdata[tid+128];
					idxData[tid] = idxData[tid+128];
				}
			}
		}
		__syncthreads();
	}

	if (blockSize >= 128)
	{
		if (tid <   64)
		{
			if(myMax < sdata[tid +   64])
			{
				if (checkDistancef(smaxesRow, smaxesCol, 
						nMax, idxData[tid+64],
						width))
				{
					sdata[tid] = myMax = sdata[tid+64];
					idxData[tid] = idxData[tid+64];
				}
			}
		}
		__syncthreads();
	}

	volatile float *vdata = sdata;
	volatile int *vidxData = idxData;
	
	volatile int *vsmaxesRow = smaxesRow;
	volatile int *vsmaxesCol = smaxesCol;

	if (tid < 32)
	{
		if (blockSize >=  64)
			if (myMax < vdata[tid + 32])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol,
						nMax, vidxData[tid+32],
						width))
				{
					vdata[tid] = myMax = vdata[tid+32];
					vidxData[tid] = vidxData[tid+32];
				}
			}

		if (blockSize >=  32)
			if (myMax < vdata[tid + 16])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+16],
						width))
				{
					vdata[tid] = myMax = vdata[tid+16];
					vidxData[tid] = vidxData[tid+16];
				}
			}

		if (blockSize >=  16)
			if (myMax < vdata[tid +  8])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+8],
						width))
				{
					vdata[tid] = myMax = vdata[tid+8];
					vidxData[tid] = vidxData[tid+8];
				}
			}

		if (blockSize >=    8)
			if (myMax < vdata[tid +  4])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+4],
						width))
				{
					vdata[tid] = myMax = vdata[tid+4];
					vidxData[tid] = vidxData[tid+4];
				}
			}

		if (blockSize >=    4)
			if (myMax < vdata[tid+2])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+2],
						width))
				{
					vdata[tid] = myMax = vdata[tid+2];
					vidxData[tid] = vidxData[tid+2];
				}
			}

		if (blockSize >=    2)
			if (myMax < vdata[tid +  1])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+1],
						width))
				{
					vdata[tid] = myMax = vdata[tid+1];
					vidxData[tid] = vidxData[tid+1];
				}

			}
		__syncthreads();
	}

	if (tid == 0)
	{
		g_odata[blockIdx.x] = sdata[0];
		max_idx[blockIdx.x] = idxData[0];
		
		if (gridDim.x == 1)		
			maxes[nMax] = idxData[0];
	}
}


extern "C"
__global__ void
reduce_max_filter_mainf(float *g_idata, float *g_odata,
		int * max_idx, unsigned int width, unsigned int height,
		int blockSize, 
		int *maxes, int nMax)
{
	__shared__ int smaxesRow[10];
	__shared__ int smaxesCol[10];
	__shared__ int smaxesVal[10];
	__shared__ float sdata[THREADS_PER_BLOCK];
	__shared__ int idxData[THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize) + tid;
	unsigned int gridSize = blockSize*gridDim.x;
	if (tid < nMax)
	{
		smaxesVal[tid] = maxes[tid];
		smaxesRow[tid] = smaxesVal[tid] / width;		
		smaxesCol[tid] = smaxesVal[tid] % width;		
	}
	__syncthreads();

	float myMax = -INFINITY;
	int myMaxIndex;
	float val;

	while (i < width * height)
	{
		val = g_idata[i];
		if (myMax < val)
		{
			// compute distance . . .
			if (checkDistancef(smaxesRow, smaxesCol, 
					nMax, i, width))
			{
				myMax = val;
				myMaxIndex = i;
			}
		}

		if (i+blockSize < width * height)
		{
			val = g_idata[i+blockSize];
			if (myMax < val)
			{

				if (checkDistancef(smaxesRow, smaxesCol, 
						nMax, i+blockSize, width))
				{
					myMax = val;
					myMaxIndex = i+blockSize;
				}
			}
		}

		i += gridSize;
	}

	sdata[tid] = myMax;
	idxData[tid] = myMaxIndex;

	__syncthreads();

	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			if (myMax < sdata[tid + 256])
			{
				if (checkDistancef(smaxesRow, smaxesCol, 
						nMax, idxData[tid+256],
						width))
				{
					sdata[tid] = myMax = sdata[tid+256];
					idxData[tid] = idxData[tid+256];
				}
			}
		}
		__syncthreads();
	}

	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			if (myMax < sdata[tid + 128])
			{
				if (checkDistancef(smaxesRow, smaxesCol, 
						nMax, idxData[tid+128],
						width))
				{
					sdata[tid] = myMax = sdata[tid+128];
					idxData[tid] = idxData[tid+128];
				}
			}
		}
		__syncthreads();
	}

	if (blockSize >= 128)
	{
		if (tid <   64)
		{
			if(myMax < sdata[tid +   64])
			{
				if (checkDistancef(smaxesRow, smaxesCol, 
						nMax, idxData[tid+64],
						width))
				{
					sdata[tid] = myMax = sdata[tid+64];
					idxData[tid] = idxData[tid+64];
				}
			}
		}
		__syncthreads();
	}

	volatile float *vdata = sdata;
	volatile int *vidxData = idxData;
	
	volatile int *vsmaxesRow = smaxesRow;
	volatile int *vsmaxesCol = smaxesCol;

	if (tid < 32)
	{
		if (blockSize >=  64)
			if (myMax < vdata[tid + 32])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+32],
						width))
				{
					vdata[tid] = myMax = vdata[tid+32];
					vidxData[tid] = vidxData[tid+32];
				}
			}

		if (blockSize >=  32)
			if (myMax < vdata[tid + 16])
			{

				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+16],
						width))
				{
					vdata[tid] = myMax = vdata[tid+16];
					vidxData[tid] = vidxData[tid+16];
				}
			}

		if (blockSize >=  16)
			if (myMax < vdata[tid +  8])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+8],
						width))
				{
					vdata[tid] = myMax = vdata[tid+8];
					vidxData[tid] = vidxData[tid+8];
				}
			}

		if (blockSize >=    8)
			if (myMax < vdata[tid +  4])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+4],
						width))
				{
					vdata[tid] = myMax = vdata[tid+4];
					vidxData[tid] = vidxData[tid+4];
				}
			}

		if (blockSize >=    4)
			if (myMax < vdata[tid+2])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+2],
						width))
				{
					vdata[tid] = myMax = vdata[tid+2];
					vidxData[tid] = vidxData[tid+2];
				}
			}

		if (blockSize >=    2)
			if (myMax < vdata[tid +  1])
			{
				if (checkDistancef(vsmaxesRow, vsmaxesCol, 
						nMax, vidxData[tid+1],
						width))
				{
					vdata[tid] = myMax = vdata[tid+1];
					vidxData[tid] = vidxData[tid+1];
				}
			}
		__syncthreads();
	}

	if (tid == 0)
	{
		g_odata[blockIdx.x] = sdata[0];
		max_idx[blockIdx.x] = idxData[0];
		
		if (gridDim.x == 1)
			maxes[nMax] = idxData[0];
		
	}
}




// ================================================================

// Local Variables:
// time-stamp-line-limit: 30
// End:

